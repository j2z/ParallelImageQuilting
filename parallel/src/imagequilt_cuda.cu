#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include "util.hpp"
#include "constants.hpp"
#include "point.hpp"
#include "color_cu.hpp"
#include "util_cu.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define NUM_ITERATIONS 20
#define BLOCK_SIZE 32

__global__ void initRandom(unsigned int seed, hiprandState* states)
{
  int idX = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  hiprand_init(seed, idX,0,&states[idX]);
}

__global__ void kernelRandomOutput(hiprandState* states, int* output, int output_width, int source_size)
{
  int idX = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  output[idX] = hiprand(&states[idX]) % source_size;
}

__global__ void kernelFindBoundaries(hiprandState* states, unsigned char* source, int sourceWidth, int sourceHeight, int* output, int xOffset, int yOffset, unsigned char* minPaths, short* samplesX, short* samplesY)
{
  int tileIdx = blockIdx.y * WIDTH_TILES + blockIdx.x;
  int idX = tileIdx * POLAR_WIDTH + threadIdx.x;

  int tileX = blockIdx.x;
  int tileY = blockIdx.y;
  int colIdx = threadIdx.x;
  
  __shared__ float array1[POLAR_WIDTH];
  __shared__ float array2[POLAR_WIDTH];
  __shared__ char back_pointers[POLAR_HEIGHT][POLAR_WIDTH];

  if (colIdx == 0)
  {
    samplesX[tileIdx] = hiprand(&states[idX]) % (sourceWidth - 2*MAX_RADIUS) + MAX_RADIUS;
    samplesY[tileIdx] = hiprand(&states[idX]) % (sourceHeight - 2*MAX_RADIUS) + MAX_RADIUS;
  }
  __syncthreads();

  int srcX = samplesX[tileIdx];
  int srcY = samplesY[tileIdx];

  int mapX = tileX * TILE_WIDTH + TILE_WIDTH / 2 + xOffset;
  int mapY = tileY * TILE_HEIGHT + TILE_HEIGHT / 2 + yOffset;

  ErrorFunctionCu errFunc(source, sourceWidth, srcX, srcY, output, OUTPUT_WIDTH, mapX, mapY, PolarTransformation(MAX_RADIUS, RADIUS_FACTOR, ANGLE_FACTOR));

  for (int i = 0; i < POLAR_HEIGHT; i++)
  {
  }
  
  

}

void imagequilt_cuda(int texture_width, int texture_height, unsigned char* source, int* output)
{
  //initialize CUDA global memory
  unsigned char* source_cuda;
  int* output_cuda;
  //actually, I think it might be possible to store the previous 2 values in shared memory
  unsigned char* min_paths;
  short* samplesX;
  short* samplesY;

  int output_height = (HEIGHT_TILES + 1)*TILE_HEIGHT;
  int output_width = (WIDTH_TILES + 1)*TILE_WIDTH;
  int tile_size = TILE_HEIGHT*TILE_WIDTH;
  int num_tiles = HEIGHT_TILES*WIDTH_TILES;

  size_t source_size = sizeof(unsigned char)*texture_width*texture_height*3;
  size_t output_size = sizeof(int)*output_width*output_height;
  size_t paths_size = sizeof(unsigned char)*POLAR_HEIGHT*num_tiles;
  size_t samples_size = sizeof(short)*num_tiles;

  hipMalloc((void**)&source_cuda, source_size);
  hipMalloc((void**)&output_cuda, output_size);
  hipMalloc((void**)&min_paths, paths_size);
  hipMalloc((void**)&samplesX, samples_size);
  hipMalloc((void**)&samplesY, samples_size);

  hipMemcpy(source_cuda, source, source_size, hipMemcpyHostToDevice);

  //first copy random pixels from source to output
  int seed = 15418;
  hiprandState *randStates;
  hipMalloc((void**)&randStates, sizeof(hiprandState)*output_width*output_height);
  int numBlocks = output_width*output_height/BLOCK_SIZE;

  initRandom<<<numBlocks, BLOCK_SIZE>>>(seed, randStates);
  kernelRandomOutput<<<numBlocks, BLOCK_SIZE>>>(randStates, 
                                                output_cuda,
                                                output_width,
                                                texture_width*texture_height);

  dim3 blockDim(POLAR_WIDTH, 1);
  dim3 gridDim(WIDTH_TILES, HEIGHT_TILES, 1);

  for (int iter = 0; iter < ITERATIONS; iter++)
  {
    //choose random grid alignment
    const int offsetX = std::rand() % TILE_WIDTH;
    const int offsetY = std::rand() % TILE_HEIGHT;

    kernelFindBoundaries<<<gridDim, blockDim>>>(randStates, texture_width, texture_height, output_cuda, offsetX, offsetY, min_paths, samplesX, samplesY);
    
  }

  hipMemcpy(output, output_cuda, output_size, hipMemcpyDeviceToHost);

  hipFree(randStates);
  hipFree(source_cuda);
  hipFree(output_cuda);
}



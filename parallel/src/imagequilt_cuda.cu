#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include "cu_helpers.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define NUM_ITERATIONS 20
#define BLOCK_SIZE 64

#define LOG2_WARP_SIZE 5U
#define WARP_SIZE (1U << LOG2_WARP_SIZE)
#define SCAN_BLOCK_DIM POLAR_WIDTH

//Almost the same as naive scan1Inclusive, but doesn't need __syncthreads()
//assuming size <= WARP_SIZE
inline __device__ uint
warpScanInclusive(int threadIndex, float idata, volatile float *s_Data, uint size){
  uint pos = 2 * threadIndex - (threadIndex & (size - 1));
  s_Data[pos] = 0;
  pos += size;
  s_Data[pos] = idata;

  for(uint offset = 1; offset < size; offset <<= 1)
    s_Data[pos] += s_Data[pos - offset];

  return s_Data[pos];
}

inline __device__ float warpScanExclusive(int threadIndex, float idata, volatile float *sScratch, uint size){
  return warpScanInclusive(threadIndex, idata, sScratch, size) - idata;
}

__inline__ __device__ void
sharedMemExclusiveScan(int threadIndex, float* sInput, float* sOutput, volatile float* sScratch, uint size)
{
  if (size > WARP_SIZE) {

    float idata = sInput[threadIndex];

    //Bottom-level inclusive warp scan
    float warpResult = warpScanInclusive(threadIndex, idata, sScratch, WARP_SIZE);

    // Save top elements of each warp for exclusive warp scan sync
    // to wait for warp scans to complete (because s_Data is being
    // overwritten)
    __syncthreads();

    if ( (threadIndex & (WARP_SIZE - 1)) == (WARP_SIZE - 1) )
      sScratch[threadIndex >> LOG2_WARP_SIZE] = warpResult;

    // wait for warp scans to complete
    __syncthreads();

    if ( threadIndex < (SCAN_BLOCK_DIM / WARP_SIZE)) {
      // grab top warp elements
      float val = sScratch[threadIndex];
      // calculate exclusive scan and write back to shared memory
      sScratch[threadIndex] = warpScanExclusive(threadIndex, val, sScratch, size >> LOG2_WARP_SIZE);
    }

    //return updated warp scans with exclusive scan results
    __syncthreads();

    sOutput[threadIndex] = warpResult + sScratch[threadIndex >> LOG2_WARP_SIZE] - idata;

    } else if (threadIndex < WARP_SIZE) {
      float idata = sInput[threadIndex];
      sOutput[threadIndex] = warpScanExclusive(threadIndex, idata, sScratch, size);
    }
}

__global__ void initRandom(unsigned int seed, hiprandState* states)
{
  int idX = (blockIdx.y*BLOCK_SIZE + blockIdx.x) * BLOCK_SIZE + threadIdx.x;
  hiprand_init(seed, idX,0,&states[idX]);
}

__global__ void kernelRandomOutput(hiprandState* states, int* output, int output_width, int source_size)
{
  int idX = (blockIdx.y*BLOCK_SIZE + blockIdx.x) * BLOCK_SIZE + threadIdx.x;
  output[idX] = hiprand(&states[idX]) % source_size;
}



__global__ void kernelFindBoundaries(hiprandState* states, unsigned char* source, int sourceWidth, int sourceHeight, int* output, int xOffset, int yOffset, char* minPaths, short* samplesX, short* samplesY)
{
  int tileIdx = blockIdx.y * WIDTH_TILES + blockIdx.x;
  int idX = tileIdx * POLAR_WIDTH + threadIdx.x;

  int tileX = blockIdx.x;
  int tileY = blockIdx.y;
  int colIdx = threadIdx.x;
  
  __shared__ float array1[POLAR_WIDTH];
  __shared__ float array2[POLAR_WIDTH];
  __shared__ char backPointers[POLAR_HEIGHT][POLAR_WIDTH];

  __shared__ float scratch[POLAR_WIDTH];
  __shared__ float existingErrors[POLAR_WIDTH];

  __shared__ MappingData mapping;

  if (colIdx == 0)
  {
    samplesX[tileIdx] = hiprand(&states[idX]) % (sourceWidth - 2*MAX_RADIUS) + MAX_RADIUS;
    samplesY[tileIdx] = hiprand(&states[idX]) % (sourceHeight - 2*MAX_RADIUS) + MAX_RADIUS;

    mapping.src = source;
    mapping.srcWidth = sourceWidth;
    mapping.srcX = samplesX[tileIdx];
    mapping.srcY = samplesY[tileIdx];
    mapping.map = output;
    mapping.mapWidth = OUTPUT_WIDTH;
    mapping.mapX = tileX * TILE_WIDTH + TILE_WIDTH / 2 + xOffset;
    mapping.mapY = tileY * TILE_HEIGHT + TILE_HEIGHT / 2 + yOffset;
  }
  __syncthreads();
  
  existingErrors[colIdx] = -existing_error(mapping, colIdx, 0);
  
  __syncthreads();

  float* currentRow = array1;
  // populates currentRow with the negative sum of existing errors
  // (not including current)
  sharedMemExclusiveScan(colIdx, existingErrors, currentRow, scratch, POLAR_WIDTH);

  currentRow[colIdx] += horiz_error(mapping, colIdx, 0) + existingErrors[colIdx];
  
  float* previousRow = currentRow;
  currentRow = array2;


  for (int theta = 1; theta < POLAR_HEIGHT; theta++)
  {
    existingErrors[colIdx] = -existing_error(mapping, colIdx, theta);

    __syncthreads();
    
    // populates currentRow with the negative sum of existing errors
    // (not including current)
    sharedMemExclusiveScan(colIdx, existingErrors, currentRow, scratch, POLAR_WIDTH);

    char minTry = -1;
    float minVal = 0.f;
    for (char arg = colIdx - 1; arg <= colIdx + 1; arg++)
    {
      if (arg >= 0 && arg < POLAR_WIDTH)
      {
        if (minTry == -1 || previousRow[arg] < minVal)
        {
          minTry = arg;
          minVal = previousRow[arg];
        }
      }
    }
    currentRow[colIdx] += minVal + horiz_error(mapping, colIdx, theta) + existingErrors[colIdx];
    backPointers[theta][colIdx] = minTry;
    
    float* temp = previousRow;
    previousRow = currentRow;
    currentRow = temp;

  }

  // at this point, previousRow stores the seam costs

  int* scratch2 = (int*)scratch;

  int index = backPointers[POLAR_HEIGHT - 1][colIdx];
  for (int step = POLAR_HEIGHT - 2; step > 0; step--)
  {
    index = backPointers[step][index];
  }
  if (index == colIdx)
  {
    scratch2[colIdx] = colIdx;
  }
  else
  {
    scratch2[colIdx] = -1;
  }

  __syncthreads();

  // do a reduction
  for (int s = 1; s < POLAR_WIDTH; s*=2)
  {
    if (colIdx % (2 * s) == 0)
    {
      if (scratch2[colIdx] == -1)
      {
        if (scratch2[colIdx + s] != -1)
        {
          scratch2[colIdx] = scratch2[colIdx + s];
          previousRow[colIdx] = previousRow[colIdx + s];
        }
      }
      else
      {
        if (scratch2[colIdx+s] != -1 &&
              previousRow[colIdx+s] < previousRow[colIdx])
        {
          scratch2[colIdx] = scratch2[colIdx + s];
          previousRow[colIdx] = previousRow[colIdx + s];
        }
      }
    }
    __syncthreads();
  }

  if (scratch2[0] == colIdx && previousRow[0] < 1.0)
  {
    char index = colIdx;
    for (int step = POLAR_HEIGHT - 1; step >= 0; step--)
    {
      minPaths[tileIdx*POLAR_HEIGHT + step] = index;
      index = backPointers[step][index];
    }
  }

}

__global__ void kernelUpdateMap(int srcWidth, int* map, int xOffset, int yOffset, char* minPaths, short* samplesX, short* samplesY);


void imagequilt_cuda(int texture_width, int texture_height, unsigned char* source, int* output)
{
  //initialize CUDA global memory
  unsigned char* source_cuda;
  int* output_cuda;
  //actually, I think it might be possible to store the previous 2 values in shared memory
  char* min_paths;
  short* samplesX;
  short* samplesY;

  int output_height = (HEIGHT_TILES + 1)*TILE_HEIGHT;
  int output_width = (WIDTH_TILES + 1)*TILE_WIDTH;
  int num_tiles = HEIGHT_TILES*WIDTH_TILES;

  size_t source_size = sizeof(unsigned char)*texture_width*texture_height*3;
  size_t output_size = sizeof(int)*output_width*output_height;
  size_t paths_size = sizeof(unsigned char)*POLAR_HEIGHT*num_tiles;
  size_t samples_size = sizeof(short)*num_tiles;

  hipMalloc((void**)&source_cuda, source_size);
  hipMalloc((void**)&output_cuda, output_size);
  hipMalloc((void**)&min_paths, paths_size);
  hipMalloc((void**)&samplesX, samples_size);
  hipMalloc((void**)&samplesY, samples_size);

  hipMemcpy(source_cuda, source, source_size, hipMemcpyHostToDevice);
  hipMemcpy(output_cuda, output, output_size, hipMemcpyHostToDevice);

  // seam carving: each tile gets 1 block of 32 threads
  dim3 seamCarveBlockDim(POLAR_WIDTH, 1);
  dim3 seamCarveGridDim(WIDTH_TILES, HEIGHT_TILES, 1);

  dim3 updateBlockDim(TILE_WIDTH / 2, TILE_HEIGHT / 2);
  dim3 updateGridDim(WIDTH_TILES, HEIGHT_TILES, 4);

  //first copy random pixels from source to output
  int seed = 15418;
  hiprandState *randStates;
  hipMalloc((void**)&randStates, sizeof(hiprandState)*num_tiles);
  
  initRandom<<<seamCarveGridDim, seamCarveBlockDim>>>(seed, randStates);
  
  for (int iter = 0; iter < ITERATIONS; iter++)
  {
    hipDeviceSynchronize();
    
    //choose random grid alignment
    const int offsetX = std::rand() % TILE_WIDTH;
    const int offsetY = std::rand() % TILE_HEIGHT;
    
    kernelFindBoundaries<<<seamCarveGridDim, seamCarveBlockDim>>>(randStates, source_cuda, texture_width, texture_height, output_cuda, offsetX, offsetY, min_paths, samplesX, samplesY);
   
    hipDeviceSynchronize();
    // activate this when ready
    
    kernelUpdateMap<<<updateGridDim, updateBlockDim>>>
      (texture_width, output_cuda, offsetX, offsetY, min_paths, samplesX, samplesY);
  }

  hipDeviceSynchronize();

  hipMemcpy(output, output_cuda, output_size, hipMemcpyDeviceToHost);

  hipFree(randStates);
  hipFree(source_cuda);
  hipFree(output_cuda);
  hipFree(min_paths);
  hipFree(samplesX);
  hipFree(samplesY);
}



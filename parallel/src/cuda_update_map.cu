#include "hip/hip_runtime.h"

#include "cu_helpers.hpp"

__global__ void kernelUpdateMap(int srcWidth, int* map, int xOffset, int yOffset, char* minPaths, bool* improvements, short* samplesX, short* samplesY)
{
  
  int tileX = blockIdx.x;
  int tileY = blockIdx.y;
  int tileIdx = tileY * WIDTH_TILES + tileX;

  if (!improvements[tileIdx])
  {
    return;
  }

  int pixelX;
  int pixelY;

  switch (blockIdx.z)
  {
    case 0:
      pixelX = threadIdx.x - TILE_WIDTH / 2;
      pixelY = threadIdx.y - TILE_HEIGHT / 2;
      break;
    case 1:
      pixelX = threadIdx.x;
      pixelY = threadIdx.y - TILE_HEIGHT / 2;
      break;
    case 2:
      pixelX = threadIdx.x - TILE_WIDTH / 2;
      pixelY = threadIdx.y;
      break;
    case 3:
      pixelX = threadIdx.x;
      pixelY = threadIdx.y;
      break;
  }

  if (pixelX * pixelX + pixelY * pixelY <= MAX_RADIUS * MAX_RADIUS)
  {
    Point polar = offsetToPolar(pixelX, pixelY);

    int theta = (int)round(polar.y) % POLAR_HEIGHT;
    int rad = (int)round(polar.x);

    char* seam = minPaths + tileIdx * POLAR_HEIGHT;

    if (rad <= seam[theta])
    {
      int imgX = samplesX[tileIdx] + pixelX;
      int imgY = samplesY[tileIdx] + pixelY;
      imgSetRef(map, OUTPUT_WIDTH,
                tileY * TILE_HEIGHT + pixelY + TILE_HEIGHT / 2 + yOffset,
                tileX * TILE_WIDTH + pixelX + TILE_WIDTH / 2 + xOffset,
                imgY * srcWidth + imgX);
    }

  }


}
